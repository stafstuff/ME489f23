/*******************************************************************************
To compile: gcc -O3 -o mandelbrot mandelbrot.c -lm
To create an image with 4096 x 4096 pixels: ./mandelbrot 4096 4096
*******************************************************************************/
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
int writeMandelbrot(const char *fileName, int width, int height, float *img, int minI, int maxI);

#define MXITER 1000

/*******************************************************************************/
// Define a complex number
typedef struct {
  double x;
  double y;
}complex_t;


/*******************************************************************************/
// Return iterations before z leaves mandelbrot set for given c
__device__ int testpoint(complex_t c){
  int iter;
  complex_t z = c;

  for(iter=0; iter<MXITER; iter++){
    // real part of z^2 + c
    double tmp = (z.x*z.x) - (z.y*z.y) + c.x;
    // update with imaginary part of z^2 + c
    z.y = z.x*z.y*2. + c.y;
    // update real part
    z.x = tmp;
    // check bound
    
    if((z.x*z.x+z.y*z.y)>4.0){ return iter;}
  }

  return iter;
}

/*******************************************************************************/
// perform Mandelbrot iteration on a grid of numbers in the complex plane
// record the  iteration counts in the count array
__global__ void mandelbrot(int Nre, int Nim, complex_t cmin, complex_t dc, float *count){
  int t= threadIdx.x;
  int b= blockIdx.x;
  int B = blockDim.x;
  int n = t + b*B;
  int nx= n%Nre;
  int ny= (n-n%Nre)/Nre;
  complex_t c;
  if (n<Nre*Nim){
    c.x=cmin.x+dc.x*nx;
    c.y=cmin.y+dc.y*ny;
    count[n] = (float )testpoint(c);
    
  }
}

/*******************************************************************************/
int main(int argc, char **argv){

  // to create a 4096x4096 pixel image
  // usage: ./mandelbrot 4096 4096
  int Nre = (argc==3) ? atoi(argv[1]): 4096;
  int Nim = (argc==3) ? atoi(argv[2]): 4096;
  // storage for the iteration counts
  float *count;
  count = (float*) malloc(Nre*Nim*sizeof(float));

  // Allocating memory to the DEVICE array
  float *count_d;
  hipMalloc(&count_d,Nre*Nim*sizeof(float));

  int T = 16*16; // number of threads per thread block
  dim3 G( (Nre*Nim+T-1)/T ); // number of thread blocks to use
  dim3 B(T);
  // Parameters for a bounding box for "c" that generates an interesting image
  // const float centRe = -.759856, centIm= .125547;
  // const float diam  = 0.151579;
  const float centRe = -0.5, centIm= 0;
  const float diam  = 3.0;

  complex_t cmin;
  complex_t cmax;
  complex_t dc;

  cmin.x = centRe - 0.5*diam;
  cmax.x = centRe + 0.5*diam;
  cmin.y = centIm - 0.5*diam;
  cmax.y = centIm + 0.5*diam;

  //set step sizes
  dc.x = (cmax.x-cmin.x)/(Nre-1);
  dc.y = (cmax.y-cmin.y)/(Nim-1);

  hipEvent_t start,end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);

  // compute mandelbrot set

  mandelbrot <<< G,B >>> (Nre,Nim,cmin,dc,count_d);

  // copy from the GPU back to the host here

  hipMemcpy(count,count_d,Nre*Nim*sizeof(float),hipMemcpyDeviceToHost);
  hipFree(count_d);
  hipEventRecord(end);
  float elapsed;
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed,start,end);
  // print elapsed time
  printf("elapsed = %f\n", (elapsed/1000));

  // output mandelbrot to ppm format image
  printf("Printing mandelbrot.ppm...");
  writeMandelbrot("mandelbrot.ppm", Nre, Nim, count, 0, 80);
  printf("done.\n");

  free(count);

  exit(0);
  return 0;
}


/* Output data as PPM file */
void saveppm(const char *filename, unsigned char *img, int width, int height){

  /* FILE pointer */
  FILE *f;

  /* Open file for writing */
  f = fopen(filename, "wb");

  /* PPM header info, including the size of the image */
  fprintf(f, "P6 %d %d %d\n", width, height, 255);

  /* Write the image data to the file - remember 3 byte per pixel */
  fwrite(img, 3, width*height, f);

  /* Make sure you close the file */
  fclose(f);
}



int writeMandelbrot(const char *fileName, int width, int height, float *img, int minI, int maxI){

  int n, m;
  unsigned char *rgb   = (unsigned char*) calloc(3*width*height, sizeof(unsigned char));

  for(n=0;n<height;++n){
    for(m=0;m<width;++m){
      int id = m+n*width;
      int I = (int) (768*sqrt((double)(img[id]-minI)/(maxI-minI)));

      // change this to change palette
      if(I<256)      rgb[3*id+2] = 255-I;
      else if(I<512) rgb[3*id+1] = 511-I;
      else if(I<768) rgb[3*id+0] = 767-I;
      else if(I<1024) rgb[3*id+0] = 1023-I;
      else if(I<1536) rgb[3*id+1] = 1535-I;
      else if(I<2048) rgb[3*id+2] = 2047-I;

    }
  }

  saveppm(fileName, rgb, width, height);

  free(rgb);
}